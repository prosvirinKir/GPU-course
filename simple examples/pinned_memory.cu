/*
This programm is an example of working with pinned memory
based on summation of two arrays:
a_i = 1/(i + 1)^2
b_i = exp(1/(i + 1))
c_i = sin(sin(a_i * b_i))
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void function(float* dA, float* dB, float* dC, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < size) {
    dC[idx] = sinf(sinf(dA[idx] + dB[idx]));
  }
}

int main() {
  float timerValueGPU, timerValueCPU;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float *hA, *hB, *hC, *dA, *dB, *dC;
  int size = 512 * 50000; // size of each array
  int N_thread = 512; // number of threads within a block
  int N_blocks;

  // create arrays on host
  unsigned int mem_size = sizeof(float) * size;
  // instead of
  // hA = (float*) malloc(mem_size);
  // hB = (float*) malloc(mem_size);
  // hC = (float*) malloc(mem_size);
  
  // we will allocate as follows
  hipHostAlloc((void**) &hA, mem_size, hipHostMallocDefault);
  hipHostAlloc((void**) &hB, mem_size, hipHostMallocDefault);
  hipHostAlloc((void**) &hC, mem_size, hipHostMallocDefault);

  // create arrays on device
  hipMalloc((void**) &dA, mem_size);
  hipMalloc((void**) &dB, mem_size);
  hipMalloc((void**) &dC, mem_size);

  // filling the arrays
  for (int i = 0; i < size; i++) {
    hA[i] = 1.0 / ((i + 1) * (i + 1));
    hB[i] = expf(1.0 / (i + 1));
    hC[i] = 0.0;
  }

  // calculating number of blocks
  if ((size % N_thread) == 0) {
    N_blocks = size / N_thread;
  }
  else {
    N_blocks = (int) (size / N_thread) + 1;
  }

  dim3 blocks(N_blocks);

  // GPU variant
  hipEventRecord(start, 0);

  hipMemcpy(dA, hA, mem_size, hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, mem_size, hipMemcpyHostToDevice);
  
  function <<< N_blocks, N_thread >>> (dA, dB, dC, size);

  hipError_t err = hipMemcpy(hC, dC, mem_size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(err));
    return -1;
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueGPU, start, stop);

  printf("\nGPU time: %f ms\n", timerValueGPU);

  // CPU variant
  hipEventRecord(start, 0);

  for (int i = 0; i < size; i++) {
    hC[i] = sinf(sinf(hA[i] + hB[i]));
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueCPU, start, stop);

  printf("\nCPU time: %f ms\n", timerValueCPU);
  printf("Rate: %f x\n", timerValueCPU / timerValueGPU);

  // memory free on host and device
  // free(hA);
  // free(hB);
  // free(hC);
  hipHostFree(hA);
  hipHostFree(hA);
  hipHostFree(hB);

  hipFree(dA);
  hipFree(dB);
  hipFree(dC);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  return 0;
}

// W/o pinned memory:
// GPU time: 64.865280 ms
// CPU time: 454.715393 ms
// Rate: 7.010151 x

// With pinned memory:
// GPU time: 25.868383 ms
// CPU time: 461.095520 ms
// Rate: 17.824675 x
