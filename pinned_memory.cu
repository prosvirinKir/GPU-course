/*
This programm is an example of working with pinned memory
based on summation of two arrays:
a_i = 1/(i + 1)^2
b_i = exp(1/(i + 1))
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void function(float* dA, float* dB, float* dC, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < size) {
    dC[idx] = dA[idx] + dB[idx];
  }
}

int main() {
  float timerValueGPU, timerValueCPU;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float *hA, *hB, *hC, *dA, *dB, *dC;
  int size = 512 * 50000; // size of each array
  int N_thread = 512; // number of threads within a block
  int N_blocks;

  // create arrays on host
  unsigned int mem_size = sizeof(float) * size;
  hA = (float*) malloc(mem_size);
  hB = (float*) malloc(mem_size);
  hC = (float*) malloc(mem_size);

  // create arrays on device
  hipMalloc((void**) &dA, mem_size);
  hipMalloc((void**) &dB, mem_size);
  hipMalloc((void**) &dC, mem_size);

  // filling the arrays
  for (int i = 0; i < size; i++) {
    hA[i] = 1.0 / ((i + 1) * (i + 1));
    hB[i] = expf(1.0 / (i + 1));
    hC[i] = 0.0;
  }

  // calculating number of blocks
  if ((size % N_thread) == 0) {
    N_blocks = size / N_thread;
  }
  else {
    N_blocks = (int) (size / N_thread) + 1;
  }

  dim3 blocks(N_blocks);

  // GPU variant
  hipEventRecord(start, 0);

  hipMemcpy(dA, hA, mem_size, hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, mem_size, hipMemcpyHostToDevice);
  
  function <<< N_blocks, N_thread >>> (dA, dB, dC, size);

  hipError_t err = hipMemcpy(hC, dC, mem_size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(err));
    return -1;
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueGPU, start, stop);

  printf("\nGPU time: %f ms\n", timerValueGPU);

  // CPU variant
  hipEventRecord(start, 0);

  for (int i = 0; i < size; i++) {
    hC[i] = hA[i] + hB[i];
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueCPU, start, stop);

  printf("\nCPU time: %f ms\n", timerValueCPU);
  printf("Rate: %f x\n", timerValueCPU / timerValueGPU);
  
  return 0;
}














